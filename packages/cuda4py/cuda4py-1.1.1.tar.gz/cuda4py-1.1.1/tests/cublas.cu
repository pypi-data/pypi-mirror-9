#include "hip/hip_runtime.h"
/*
 * See 7_CUDALibraries/simpleDevLibCUBLAS/ in NVIDIA CUDA Samples for more details.
 */
#include <hipblas.h>

extern "C"
__global__ void test(int matrix_side,
                     const float *alpha,
                     const float *A,
                     const float *B,
                     const float *beta,
                     float *C) {
  hipblasHandle_t blas;
  hipblasCreate(&blas); 

  hipblasSgemm(blas, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_side, matrix_side, matrix_side,
              alpha, A, matrix_side, B, matrix_side, beta, C, matrix_side);

  hipblasDestroy(blas);
}
