#include "hip/hip_runtime.h"
#include <hipblas.h>


extern "C"
__global__ void create_cublas(hipblasHandle_t *pBlas) {
  hipblasCreate(pBlas);
}


extern "C"
__global__ void destroy_cublas(const hipblasHandle_t blas) {
  hipblasDestroy(blas);
}


extern "C"
__global__ void test(const int N,
                     const hipblasHandle_t blas,
                     const int matrix_side,
                     const float *alpha,
                     const float *A,
                     const float *B,
                     const float *beta,
                     float *C) {
  for (int i = 0; i < N; i++) {
    hipblasSgemm(blas, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_side, matrix_side, matrix_side,
                alpha, A, matrix_side, B, matrix_side, beta, C, matrix_side);
  }
}


extern "C"
__global__ void test_full(
                     const int N,
                     const int matrix_side,
                     const float *alpha,
                     const float *A,
                     const float *B,
                     const float *beta,
                     float *C) {
  for (int i = 0; i < N; i++) {
    hipblasHandle_t blas;
    hipblasCreate(&blas);
    hipblasSgemm(blas, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_side, matrix_side, matrix_side,
                alpha, A, matrix_side, B, matrix_side, beta, C, matrix_side);
    hipblasDestroy(blas);
  }
}


extern "C"
__global__ void dummy(const float *in, float *out) {
  size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  out[idx] = in[idx];
}
